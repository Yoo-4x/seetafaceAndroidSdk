#include "hip/hip_runtime.h"
#include <kernels/gpu/div.h>
#include <core/tensor_builder.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <global/operator_factory.h>
#include <global/fp16_operator_factory.h>
#include <core/device.h>

#include <numeric>
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void reduce_operator_scalar_kernel(T* data, int size, const T *scalar, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] = (*scalar) == T(0)
                ? (data[index] > T(0) ? maxvalue : minvalue)
                : data[index] / (*scalar);
            }
        }

        template<typename T>
        static __global__ void reduce_operator_scalar_cross_kernel(T* data, int size, const T *scalar, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] = data[index] == T(0)
                              ? ((*scalar) > T(0) ? maxvalue : minvalue)
                              : (*scalar) / data[index];
            }
        }

        template<typename T>
        static __global__ void reduce_operator_same_shape_kernel(T* data, const T*bias, int size, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] = (bias[index]) == T(0)
                ? (data[index] > T(0) ? maxvalue : minvalue)
                : data[index] / (bias[index]);
            }
        }

        template<typename T>
        static __global__ void reduce_operator_bias_kernel(T* data, int size, int step, int slice,
                                        const T* bias, int biaslen, T maxvalue, T minvalue ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                data[index] = (bias[dim]) == T(0)
                ? (data[index] > T(0) ? maxvalue: minvalue)
                : data[index] / (bias[dim]);
            }
        }

        template<typename T>
        static __global__ void reduce_operator_bias_cross_kernel(T* data, int size, int step, int slice,
                                                           const T* bias, int biaslen, T maxvalue, T minvalue ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % (step * slice) / (step);
                data[index] = (data[index]) == T(0)
                              ? (bias[dim] > T(0) ? maxvalue : minvalue)
                              : bias[dim] / (data[index]);
            }
        }


        template<typename T>
        static __global__ void reduce_operator_kernel(T* out, int size, const T* lhs,  const T* rhs,
                                               int *lhsshape, int *lhsweight,
                                               int *rhsshape, int *rhsweight,
                                               int *outweight, int shapelen, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= size)
                return;

            int *ptmp = outweight + 1;
            int ntmp = index;

            int rhsindex = 0;
            int lhsindex = 0;
            int nbuff1,nbuff2;
            nbuff1 = nbuff2 = 0;
            for(int m = 0, i= shapelen - 1; i >= 0; --i, m++) {
                if(i > 0) {
                    nbuff1 = ntmp / *ptmp;
                    ntmp %= *ptmp;
                }else {
                    nbuff1 = ntmp;
                }

                nbuff2 = nbuff1 % lhsshape[m];
                if(m < shapelen - 1) {
                    lhsindex += nbuff2 * lhsweight[m+1];
                }else {
                    lhsindex += nbuff2;
                }

                nbuff2 = nbuff1 % rhsshape[m];

                if(m < shapelen - 1) {
                    rhsindex += nbuff2 * rhsweight[m+1];
                }else {
                    rhsindex += nbuff2;
                }

                ++ptmp;
            }

            out[index] = (rhs[rhsindex]) == T(0)
                ? (lhs[lhsindex] > T(0) ? maxvalue : minvalue)
                : lhs[lhsindex] / (rhs[rhsindex]);

        }

#ifdef TS_USE_CUDA_FP16
        template<>
        __global__ void reduce_operator_scalar_kernel<half>(half* data, int size, const half *scalar, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero = half(0.f);
            if (index < size) {
                data[index] = (*scalar) == zero
                              ? (data[index] > zero ? maxvalue : minvalue)
                              : data[index] / (*scalar);
            }
        }

        template<>
        __global__ void reduce_operator_scalar_cross_kernel<half>(half* data, int size, const half *scalar, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero = half(0.f);
            if (index < size) {
                data[index] = data[index] == zero
                              ? ((*scalar) > zero ? maxvalue : minvalue)
                              : (*scalar) / data[index];
            }
        }

        template<>
        __global__ void reduce_operator_same_shape_kernel<half>(half* data, const half* bias, int size, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero = half(0.f);
            if (index < size) {
                data[index] = (bias[index]) == zero
                              ? (data[index] > zero ? maxvalue : minvalue)
                              : data[index] / (bias[index]);
            }
        }

        template<>
        __global__ void reduce_operator_bias_kernel<half>(half* data, int size, int step, int slice,
                                                          const half* bias, int biaslen, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero = half(0.f);
            if (index < size) {
                int dim = index % (step * slice) / (step);
                data[index] = (bias[dim]) == zero
                              ? (data[index] > zero ? maxvalue : minvalue)
                              : data[index] / (bias[dim]);
            }
        }

        template<>
        __global__ void reduce_operator_bias_cross_kernel<half>(half* data, int size, int step, int slice,
                                                                const half* bias, int biaslen, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero = half(0.f);
            if (index < size) {
                int dim = index % (step * slice) / (step);
                data[index] = (data[index]) == zero
                              ? (bias[dim] > zero ? maxvalue : minvalue)
                              : bias[dim] / (data[index]);
            }
        }

        template<>
        __global__ void reduce_operator_kernel<half>(half* out, int size, const half* lhs, const half* rhs,
            int *lhsshape, int *lhsweight,
            int *rhsshape, int *rhsweight,
            int *outweight, int shapelen, half maxvalue, half minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= size)
                return;

            half zero = half(0.f);

            int *ptmp = outweight + 1;
            int ntmp = index;

            int rhsindex = 0;
            int lhsindex = 0;
            int nbuff1, nbuff2;
            nbuff1 = nbuff2 = 0;
            for (int m = 0, i = shapelen - 1; i >= 0; --i, m++) {
                if (i > 0) {
                    nbuff1 = ntmp / *ptmp;
                    ntmp %= *ptmp;
                }
                else {
                    nbuff1 = ntmp;
                }

                nbuff2 = nbuff1 % lhsshape[m];
                if (m < shapelen - 1) {
                    lhsindex += nbuff2 * lhsweight[m + 1];
                }
                else {
                    lhsindex += nbuff2;
                }

                nbuff2 = nbuff1 % rhsshape[m];

                if (m < shapelen - 1) {
                    rhsindex += nbuff2 * rhsweight[m + 1];
                }
                else {
                    rhsindex += nbuff2;
                }

                ++ptmp;
            }

            out[index] = (rhs[rhsindex]) == zero
                ? (lhs[lhsindex] > zero ? maxvalue : minvalue)
                : lhs[lhsindex] / (rhs[rhsindex]);

        }
#endif


        template<typename T>
        static inline void div_gpu_compute_run(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            HypeShape lhs_hype(lhs.sizes());
            HypeShape rhs_hype(rhs.sizes());
            HypeShape out_hype(out.sizes());

            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto ncount = out.count();

            int *lhsshape = nullptr;
            int *rhsshape = nullptr;
            int *lhsweight = nullptr;
            int *rhsweight = nullptr;
            int *outweight = nullptr;

            /////////////////////////////////////
            Shape tmpshape;
            tmpshape.resize(1);
            tmpshape[0] = int32_t(lhs.sizes().size());
            Tensor lhs_tensor(out.device(), INT32, tmpshape);
            lhsshape = lhs_tensor.data<int32_t>();

            tmpshape[0] = int32_t(rhs.sizes().size());
            Tensor rhs_tensor(out.device(), INT32, tmpshape);
            rhsshape = rhs_tensor.data<int32_t>();

            tmpshape[0] = int32_t(lhs.sizes().size());
            Tensor lhs_weight_tensor(out.device(), INT32, tmpshape);
            lhsweight = lhs_weight_tensor.data<int32_t>();

            tmpshape[0] = int32_t(rhs.sizes().size());
            Tensor rhs_weight_tensor(out.device(), INT32, tmpshape);
            rhsweight = rhs_weight_tensor.data<int32_t>();

            tmpshape[0] = int32_t(out.sizes().size());
            Tensor out_weight_tensor(out.device(), INT32, tmpshape);
            outweight = out_weight_tensor.data<int32_t>();


            memcpy((void*)lhsshape, out.device(), lhs.sizes().size() * sizeof(int32_t),
                   (void*)lhs.sizes().data(), MemoryDevice(CPU), lhs.sizes().size() * sizeof(int32_t));

            memcpy((void*)rhsshape, out.device(), rhs.sizes().size() * sizeof(int32_t),
                   (void*)rhs.sizes().data(), MemoryDevice(CPU), rhs.sizes().size() * sizeof(int32_t));

            memcpy((void*)lhsweight, out.device(), lhs_hype.weight().size() * sizeof(int32_t),
                   (void*)lhs_hype.weight().data(), MemoryDevice(CPU), lhs_hype.weight().size() * sizeof(int32_t));

            memcpy((void*)rhsweight, out.device(), rhs_hype.weight().size() * sizeof(int32_t),
                   (void*)rhs_hype.weight().data(), MemoryDevice(CPU), rhs_hype.weight().size() * sizeof(int32_t));
            memcpy((void*)outweight, out.device(), out_hype.weight().size() * sizeof(int32_t),
                   (void*)out_hype.weight().data(), MemoryDevice(CPU), out_hype.weight().size() * sizeof(int32_t));
            /////////////////////////////////////

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();

            RUN_KERNEL(reduce_operator_kernel<T>, CUDA_BLOCK(ncount, CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, ncount, plhs, prhs, lhsshape, lhsweight, rhsshape, rhsweight, outweight,
                       int(out.sizes().size()), maxvalue, minvalue);

        }


        template<typename T>
        static inline void div_gpu_compute_run_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            memcpy((void*)pout, out.device(), out.count() * sizeof(T),
                   (void*)plhs, lhs.device(), out.count() * sizeof(T));

            RUN_KERNEL(reduce_operator_scalar_kernel<T>, CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, out.count(), prhs, maxvalue, minvalue);

        }

        template<typename T>
        static inline void div_gpu_compute_run_scalar_cross(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            memcpy((void*)pout, out.device(), out.count() * sizeof(T),
                   (void*)prhs, rhs.device(), out.count() * sizeof(T));

            RUN_KERNEL(reduce_operator_scalar_cross_kernel<T>,
                       CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, out.count(), plhs, maxvalue, minvalue);
        }


        template<typename T>
        static inline void div_gpu_compute_run_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();

            memcpy((void*)pout, out.device(), out.count() * sizeof(T),
                   (void*)plhs, lhs.device(), out.count() * sizeof(T));

            RUN_KERNEL(reduce_operator_same_shape_kernel<T>,
                       CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, prhs, out.count(), maxvalue, minvalue);
        }


        template<typename T>
        static inline void div_gpu_compute_run_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto &out_shape = out.sizes();
            auto number = std::accumulate(out_shape.begin(), out_shape.begin() + dim, 1, std::multiplies<int>());
            auto count = std::accumulate(out_shape.begin() + dim + 1, out_shape.end(), 1, std::multiplies<int>());

            auto channels = out_shape[dim];

            memcpy((void*)pout, out.device(), out.count() * sizeof(T),
                   (void*)plhs, lhs.device(), out.count() * sizeof(T));

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            RUN_KERNEL(reduce_operator_bias_kernel<T>, CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, out.count(),
                       count, channels, prhs, rhs.count(), maxvalue, minvalue);

        }

        template<typename T>
        static inline void div_gpu_compute_run_bias_cross(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto &out_shape = out.sizes();
            auto number = std::accumulate(out_shape.begin(), out_shape.begin() + dim, 1, std::multiplies<int>());
            auto count = std::accumulate(out_shape.begin() + dim + 1, out_shape.end(), 1, std::multiplies<int>());
            auto channels = out_shape[dim];

            memcpy((void*)pout, out.device(), out.count() * sizeof(T),
                   (void*)prhs, rhs.device(), out.count() * sizeof(T));

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            RUN_KERNEL(reduce_operator_bias_cross_kernel<T>,
                       CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pout, out.count(), count, channels, plhs, lhs.count(), maxvalue, minvalue);

        }


        void Div::reduce_with_broadcast(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_scalar<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_bias<TYPE>(lhs, rhs, out, dim); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_same_shape<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_scalar_cross(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_scalar_cross<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_bias_cross(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_bias_cross<TYPE>(lhs, rhs, out, dim); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(Div, GPU, name::layer::div())
#ifdef TS_USE_CUDA_FP16
TS_REGISTER_FP16_OPERATOR(Div, ts::GPU, name::layer::div())
#endif
