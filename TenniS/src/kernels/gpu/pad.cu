#include "hip/hip_runtime.h"
#include <kernels/gpu/pad.h>
#include <core/tensor_builder.h>
#include <global/operator_factory.h>
#include <global/fp16_operator_factory.h>
#include <backend/name.h>
#include <core/device.h>
#include <utils/assert.h>

#include <core/memory.h>
#include <numeric>

#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {
        template <typename T>
        static __global__ void pad_gpu_kernel(int count, const int *padding, T padding_value, const T *in, T *out, GpuHypeShape in_shape, GpuHypeShape out_shape) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= count) return;

            int out_index = index;
            int in_index = 0;

            auto out_weight_it = out_shape.weights + 1;
            auto in_weight_it = in_shape.weights + 1;
            /* ============================================ */
            auto in_shape_it = in_shape.shape;
            auto padding_it = padding;
            /* -------------------------------------------- */

            for (int times = out_shape.dims - 1; times; --times) {
                auto coord = index / *out_weight_it;
                /* ============================================ */
                coord -= *padding_it;
                if (coord < 0 || coord >= *in_shape_it) {
                    out[out_index] = padding_value;
                    return;
                }
                ++in_shape_it;
                padding_it += 2;
                /* -------------------------------------------- */
                in_index += coord * *in_weight_it;
                index %= *out_weight_it;
                ++out_weight_it;
                ++in_weight_it;
            }
            auto coord = index;
            /* ============================================ */
            coord -= *padding_it;
            if (coord < 0 || coord >= *in_shape_it) {
                out[out_index] = padding_value;
                return;
            }
            /* -------------------------------------------- */
            in_index += coord;

            /* ++++++++++++++++++++++++++++++++++++++++++++ */
            out[out_index] = in[in_index];
        }

        template <typename T>
        static inline void pad_gpu_compute_run(const Tensor &x, const std::vector<std::array<int, 2>> &padding, float padding_value, Tensor &out) {
            int *gpu_padding = nullptr;
            auto gpu_memory = MakeGPUHypeShape(out.device(), {x.sizes(), out.sizes()},
                                               {{(void *) (padding.data()), int(sizeof(int) * padding.size()) * 2}},
                                               {(void **) (&gpu_padding)});
            auto &gpu_in_shape = gpu_memory.second[0];
            auto &gpu_out_shape = gpu_memory.second[1];
            auto in_data = x.data<T>();
            auto out_data = out.data<T>();
            auto count = out.count();

            RUN_KERNEL(pad_gpu_kernel<T>, CUDA_BLOCK(count, CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       count, gpu_padding, T(padding_value), in_data, out_data, gpu_in_shape, gpu_out_shape);
        }

        void PadOnGPU::pad(const Tensor &x, const std::vector<std::array<int, 2>> &padding, float padding_value, Tensor &out) {
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { pad_gpu_compute_run<TYPE>(x, padding, padding_value, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(PadOnGPU, GPU, name::layer::pad())
#ifdef TS_USE_CUDA_FP16
TS_REGISTER_FP16_OPERATOR(PadOnGPU, GPU, name::layer::pad())
#endif
