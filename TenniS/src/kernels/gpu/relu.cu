#include "hip/hip_runtime.h"
#include <kernels/gpu/relu.h>
#include <algorithm>

#include "backend/name.h"
#include "global/operator_factory.h"
#include "global/fp16_operator_factory.h"
#include "kernels/gpu/memory_gpu.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {

        template<typename T>
        __global__ static void relu_kernel(const T* input_data, T* output_data, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size)
            {
                T val = input_data[index];
                output_data[index] = val > T(0.0) ? val : T(0.0);
            }
        }

#ifdef TS_USE_CUDA_FP16
        template<>
        __global__ void relu_kernel<half>(const half* input_data, half* output_data, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero(0.f);
            if (index < size)
            {
                half val = input_data[index];
                output_data[index] = val > zero ? val : zero;
            }
        }
#endif

        template<typename T>
        void cpu_relu_compute_run(const Tensor &x, Tensor &out) {
            const T *input_data = x.data<T>();
            T *output_data = out.data<T>();
            int count = out.count();
            // int bytes_num = count * sizeof(T);

            dim3 blockSize(CUDA_THREAD_NUM);
            dim3 gridSize(CUDA_BLOCK(count, blockSize.x));

            RUN_KERNEL(relu_kernel<T>, gridSize, blockSize, input_data, output_data, count);
        }

        void ReLU::active(const Tensor &x, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { cpu_relu_compute_run<TYPE>(x, out); break; }
                //DECLARE_COMPUTE_RUN(INT8, int8_t);
                //DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                //DECLARE_COMPUTE_RUN(INT16, int16_t);
                //DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                //DECLARE_COMPUTE_RUN(INT32, int32_t);
                //DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                //DECLARE_COMPUTE_RUN(INT64, int64_t);
                //DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                break;
            }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(ReLU, ts::GPU, name::layer::relu())
TS_REGISTER_FP16_OPERATOR(ReLU, ts::GPU, name::layer::relu())