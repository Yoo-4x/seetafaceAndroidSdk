#include "hip/hip_runtime.h"
#include <kernels/gpu/depthwise_conv2d_core.h>
#include <core/tensor_builder.h>
#include <backend/name.h>
#include <utils/assert.h>

#include <kernels/gpu/operator_on_gpu.h>
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {

        template <typename T>
        static __global__ void gpu_depthwise_conv2d_nchw_kernel(
                                int nthreads, const T* bottom_data, const T* weight_data, 
                                int num, int channels,
                                int top_height, int top_width, int bottom_height, int bottom_width,
                                int kernel_h,  int kernel_w,  int stride_h, int stride_w,
                                int pad_top, int pad_bottom, int pad_left, int pad_right, 
                                int dilation_h, int dilation_w, T* top_data) {

            for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads; index += blockDim.x * gridDim.x) {
                const int n = index / channels / top_height / top_width;
                const int c = (index / top_height / top_width) % channels;
                const int h = (index / top_width) % top_height;
                const int w = index % top_width;
                const T* weight = weight_data + c * kernel_h * kernel_w;
                T value = 0;
                for (int kh = 0; kh < kernel_h; ++kh) {
                    for (int kw = 0; kw < kernel_w; ++kw) {
                        const int h_in = -pad_top + h * stride_h + kh * dilation_h;
                        const int w_in = -pad_left + w * stride_w + kw * dilation_w;
                        if ((h_in >= 0) && (h_in < bottom_height) && (w_in >= 0) && (w_in < bottom_width)) {
                            const int offset = ((n * channels + c) * bottom_height + h_in) * bottom_width + w_in;
                            value += (*weight) * bottom_data[offset];
                        }
                        ++weight;
                    }
                }
                top_data[index] = value;
            }
        }

#ifdef TS_USE_CUDA_FP16
        template <>
        __global__ void gpu_depthwise_conv2d_nchw_kernel<half>(
            int nthreads, const half* bottom_data, const half* weight_data,
            int num, int channels,
            int top_height, int top_width, int bottom_height, int bottom_width,
            int kernel_h, int kernel_w, int stride_h, int stride_w,
            int pad_top, int pad_bottom, int pad_left, int pad_right,
            int dilation_h, int dilation_w, half* top_data) {

            for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads; index += blockDim.x * gridDim.x) {
                const int n = index / channels / top_height / top_width;
                const int c = (index / top_height / top_width) % channels;
                const int h = (index / top_width) % top_height;
                const int w = index % top_width;
                const half* weight = weight_data + c * kernel_h * kernel_w;
                half value = half(0.f);
                for (int kh = 0; kh < kernel_h; ++kh) {
                    for (int kw = 0; kw < kernel_w; ++kw) {
                        const int h_in = -pad_top + h * stride_h + kh * dilation_h;
                        const int w_in = -pad_left + w * stride_w + kw * dilation_w;
                        if ((h_in >= 0) && (h_in < bottom_height) && (w_in >= 0) && (w_in < bottom_width)) {
                            const int offset = ((n * channels + c) * bottom_height + h_in) * bottom_width + w_in;
                            value = value + (*weight) * bottom_data[offset];
                        }
                        ++weight;
                    }
                }
                top_data[index] = value;
            }
        }
#endif

        template<typename T>
        static void gpu_depthwise_conv2d_nchw_compute_run(const Tensor &x, const Padding2D &padding, float padding_value,
                                              const Tensor &weight, const Stride2D &stride, const Dilation2D &dilation,
                                              Tensor &out, Stack &stack) {
            auto weight_shape = weight.sizes();
            auto output_shape = out.sizes();
            auto input_shape = x.sizes();

            const T *pinput = x.data<T>();
            const T *pweight_base = weight.data<T>();
            T *poutput = out.data<T>();

            RUN_KERNEL(gpu_depthwise_conv2d_nchw_kernel<T>, CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       out.count(), pinput, pweight_base, output_shape[0], output_shape[1],
                       output_shape[2], output_shape[3], input_shape[2], input_shape[3],
                       weight_shape[2], weight_shape[3], stride.height, stride.width,
                       padding.top, padding.bottom, padding.left, padding.right,
                       dilation.height, dilation.width, poutput);
        }

        void
        DepthwiseConv2DCore::conv2d(const Tensor &x, const Padding2D &padding, float padding_value, const Tensor &w,
                                    const Stride2D &stride, const Dilation2D &dilation, Conv2DFormat format,
                                    Tensor &out, Stack &stack) {
            if (format != FORMAT_NCHW) {
                TS_LOG_ERROR << "DepthwiseConv2DCore Conv2D only support NCHW" << eject;
            }
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_depthwise_conv2d_nchw_compute_run<TYPE>(x, padding, padding_value, w, stride, dilation, out, stack);; break; }
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "DepthwiseConv2DCore not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }
    }
}
