#include "hip/hip_runtime.h"
//
// Created by kier on 2019/3/6.
//

#include "kernels/gpu/gatherv2.h"
#include "global/operator_factory.h"
#include "backend/name.h"
#include "kernels/gpu/gpu_kernel.h"
#include <numeric>
#include <hip/hip_runtime.h>
#include <>

#include "kernels/gpu/cuda_context.h"
#include "core/device_context.h"
#include "utils/ctxmgr_lite.h"

namespace ts {
    namespace gpu {
        static __global__ void gpu_gatherv2_kernel(int count, const char * x_data, const int * indices_data, char * out_data, 
                                                   int axis, int bytes, int width_bytes, GpuHypeShape c_shape) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= count) return;

            int in_index = 0;
            auto in_weight_it = c_shape.weights + 1;
            int curindex = 0;
            
            for (int  k=0; k < axis; k++) {
                curindex = indices_data[index * axis + k];

                if(k >= c_shape.dims -1) {
                    in_index += curindex;
                }else {
                    in_index += *in_weight_it  * curindex;
                    in_weight_it++;
                }
            }

            auto src_ptr = x_data + in_index * bytes;
            auto dst_ptr = out_data + index * width_bytes;
            ::memcpy((void *)dst_ptr, (void *)src_ptr,width_bytes);
        }


        void GatherV2::gather(const Tensor &x, const Tensor &indices, Tensor &out) {
            auto memcpy_handler = HardConverter::Query(out.device().type(), x.device().type());
            TS_AUTO_CHECK(memcpy_handler != nullptr);
            auto device_id = out.device().id();

            auto &x_shape = x.sizes();
            auto &i_shape = indices.sizes();

            int axis = i_shape[i_shape.size() - 1];

            auto number = std::accumulate(i_shape.begin(), i_shape.end() - 1, 1, std::multiplies<int>());
            auto width = std::accumulate(x_shape.begin() + axis, x_shape.end(), 1, std::multiplies<int>());

            auto gpu_hype_shape = MakeGPUHypeShape(x.device(), {x_shape});
            auto &x_hype_shape = gpu_hype_shape.second[0];

            auto bytes = x.proto().type_bytes();
            auto width_bytes = width * bytes;

            auto x_data = x.data<char>();
            auto out_data = out.data<char>();
            auto indices_data = indices.data<int32_t>();

            RUN_KERNEL(gpu_gatherv2_kernel, CUDA_BLOCK(number, CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       number, x_data, indices_data, out_data, axis, bytes, width_bytes, x_hype_shape);

        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(GatherV2, GPU, name::layer::gatherv2())
