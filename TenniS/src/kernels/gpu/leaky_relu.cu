#include "hip/hip_runtime.h"
#include <backend/base/base_leaky_relu.h>
#include "kernels/gpu/operator_on_gpu.h"

#include "backend/name.h"
#include "global/operator_factory.h"
#include "global/fp16_operator_factory.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>

#include "kernels/gpu/gpu_kernel.h"
//#include <thrust/functional.h>

namespace ts {
    namespace gpu {
        class LeakyReLU : public OperatorOnGPU<base::LeakyReLU> {
        public:
            using self = LeakyReLU;
            using supper = OperatorOnGPU<base::LeakyReLU>;

            void leaky_relu(const Tensor &x, float scale, Tensor &out) override;
        };

        template<typename T>
        __global__ static void leaky_relu_kernel(const T* input_data, T* output_data,T scale, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            //thrust::maximum<T> mx;
            //thrust::minimum<T> mn;
            if (index < size)
            {
                //T val = input_data[index];
                //output_data[index] = mn(mx(val, T(0), max));
                T val = input_data[index];
                output_data[index] = val > T(0) ? val : val * scale;
            }
        }

#ifdef TS_USE_CUDA_FP16
        template<>
        __global__ void leaky_relu_kernel<half>(const half* input_data, half* output_data, half scale, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            half zero(0.f);
            if (index < size)
            {
                half val = input_data[index];
                half max_temp = val > zero ? val : zero;
                half min_temp = val < zero ? val : zero;
                output_data[index] = max_temp + scale * min_temp;
            }
        }
#endif

        template<typename T>
        static void gpu_leaky_relu_compute_run(const Tensor &x, float scale, Tensor &out) {
            const T *input_data = x.data<T>();
            T *output_data = out.data<T>();
            int count = out.count();

            T casted_scale = T(scale);
            
            dim3 blockSize(CUDA_THREAD_NUM);
            dim3 gridSize(CUDA_BLOCK(count, blockSize.x));

            RUN_KERNEL(leaky_relu_kernel<T>, gridSize, blockSize, input_data, output_data, casted_scale, count);
        }

        void LeakyReLU::leaky_relu(const Tensor &x, float scale, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_leaky_relu_compute_run<TYPE>(x, scale, out); break; }
                //DECLARE_COMPUTE_RUN(INT8, int8_t);
                //DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                //DECLARE_COMPUTE_RUN(INT16, int16_t);
                //DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                //DECLARE_COMPUTE_RUN(INT32, int32_t);
                //DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                //DECLARE_COMPUTE_RUN(INT64, int64_t);
                //DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                break;
            }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(LeakyReLU, ts::GPU, "leaky_relu")
#ifdef TS_USE_CUDA_FP16
TS_REGISTER_FP16_OPERATOR(LeakyReLU, ts::GPU, "leaky_relu")
#endif
