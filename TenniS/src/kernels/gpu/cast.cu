#include "hip/hip_runtime.h"
#include <kernels/gpu/cast.h>
#include <core/tensor_builder.h>
#include <global/operator_factory.h>
#include "global/fp16_operator_factory.h"
#include <backend/name.h>
#include <utils/assert.h>
#include <core/device.h>

#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

/////////////////////////////////////////////////
namespace ts {
    namespace gpu {
        template<typename T_IN, typename T_OUT>
        static __global__ void gpu_cast_kernel(T_OUT * dst, const T_IN * src, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                dst[index] = static_cast<T_OUT>(src[index]);
            }
        }

        template<typename T_IN>
        static __global__ void gpu_cast_kernel(half * dst, const T_IN * src, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                dst[index] = __float2half(static_cast<float>(src[index]));
            }
        }

        template<typename T_OUT>
        static __global__ void gpu_cast_kernel(T_OUT * dst, const half * src, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                dst[index] = static_cast<T_OUT>(__half2float(src[index]));
            }
        }

        static __global__ void gpu_cast_kernel(half * dst, const half * src, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                dst[index] = src[index];
            }
        }

        template<typename T_IN, typename T_OUT>
        static void gpu_cast_compute_run_template(const Tensor &x, Tensor &out) {

            const T_IN *psrc = x.data<T_IN>();
            T_OUT *pdst = out.data<T_OUT>();

            if (x.dtype() == out.dtype()) {
                memcpy((void*)pdst, out.device(), x.count() * sizeof(T_IN),
                    (void*)psrc, x.device(), x.count() * sizeof(T_IN));

                return;
            }

            RUN_KERNEL(gpu_cast_kernel, CUDA_BLOCK(x.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       pdst, psrc, x.count());

        }

        template<typename T_IN>
        static void gpu_cast_compute_run(const Tensor &x, DTYPE to_type, Tensor &out) {
            switch (to_type) {
#define DECLARE_COMPUTE_RUN_TEMPLATE(DTYPE, TYPE) \
        case DTYPE: { gpu_cast_compute_run_template<T_IN, TYPE>(x, out); break; }
                DECLARE_COMPUTE_RUN_TEMPLATE(INT8, int8_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(INT16, int16_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(INT32, int32_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(INT64, int64_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN_TEMPLATE(FLOAT32, float);
                DECLARE_COMPUTE_RUN_TEMPLATE(FLOAT64, double);
                DECLARE_COMPUTE_RUN_TEMPLATE(FLOAT16, half);
                DECLARE_COMPUTE_RUN_TEMPLATE(BOOLEAN, uint8_t);
#undef DECLARE_COMPUTE_RUN_TEMPLATE
            default: {
                TS_LOG_ERROR << "_cast not support data type(" << to_type << "): " << type_str(to_type) << eject;
                break;
            }
            }
        }



        void CastV2::cast(const Tensor &x, DTYPE to_type, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = x.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_cast_compute_run<TYPE>(x, to_type, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
                DECLARE_COMPUTE_RUN(FLOAT16, half);
                DECLARE_COMPUTE_RUN(BOOLEAN, uint8_t);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                break;
            }
            }
        }

    }
}
/////////////////////////////////////////////////

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(CastV2, GPU, name::layer::cast())
TS_REGISTER_FP16_OPERATOR(CastV2, ts::GPU, name::layer::cast())
