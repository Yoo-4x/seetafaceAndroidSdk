#include "hip/hip_runtime.h"
#include <kernels/gpu/dimshuffle.h>
#include <core/tensor_builder.h>
#include <set>
#include <global/operator_factory.h>
#include <global/fp16_operator_factory.h>
#include <backend/name.h>
#include <core/device.h>
#include <utils/assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {
        template <typename T>
        static __global__ void gpu_dimshuffle_kernel(int count, const T* in, GpuHypeShape in_shape, T *out, GpuHypeShape out_shape, int dim, int *shuffle) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= count) return;

            int out_index = index;
            int in_index = 0;

            auto out_weight_it = out_shape.weights + 1;
            auto in_weight_it = in_shape.weights + 1;
            /* ============================================ */
            int running_dim = 0;
            /* -------------------------------------------- */

            for (int times = out_shape.dims - 1; times; --times) {
                auto coord = index / *out_weight_it;
                /* ============================================ */
                if (running_dim == dim) coord = shuffle[coord];
                ++running_dim;
                /* -------------------------------------------- */
                in_index += coord * *in_weight_it;
                index %= *out_weight_it;
                ++out_weight_it;
                ++in_weight_it;
            }
            auto coord = index;
            /* ============================================ */
            if (running_dim == dim) coord = shuffle[coord];
            /* -------------------------------------------- */
            in_index += coord;

            /* ++++++++++++++++++++++++++++++++++++++++++++ */
            out[out_index] = in[in_index];
        }

        template <typename T>
        static void gpu_dimshuffle_comput_run(const Tensor &x, int dim, const std::vector<int> &shuffle, Tensor &out) {
            int *gpu_shuffle = nullptr;
            auto gpu_memory = MakeGPUHypeShape(out.device(), {x.sizes(), out.sizes()},
                                               {{(void *) (shuffle.data()), int(sizeof(int) * shuffle.size())}},
                                               {(void **) (&gpu_shuffle)});
            auto &gpu_in_shape = gpu_memory.second[0];
            auto &gpu_out_shape = gpu_memory.second[1];
            auto in_data = x.data<T>();
            auto out_data = out.data<T>();
            auto count = out.count();

            RUN_KERNEL(gpu_dimshuffle_kernel<T>, CUDA_BLOCK(count, CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       count, in_data, gpu_in_shape, out_data, gpu_out_shape, dim, gpu_shuffle);
        }

        void Dimshuffle::dimshuffle(const Tensor &x, int dim, const std::vector<int> &shuffle, Tensor &out) {
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_dimshuffle_comput_run<TYPE>(x, dim, shuffle, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT16, half);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }
    }
}

///////////////////////////////////////////
using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(Dimshuffle, GPU, name::layer::dimshuffle())
TS_REGISTER_FP16_OPERATOR(Dimshuffle, ts::GPU, name::layer::dimshuffle())
