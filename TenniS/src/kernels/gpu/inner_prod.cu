#include "hip/hip_runtime.h"
#include <kernels/gpu/inner_prod.h>
#include <core/tensor_builder.h>
#include <kernels/cpu/math_cpu.h>
#include <global/operator_factory.h>
#include <global/fp16_operator_factory.h>
#include <backend/name.h>
#include <core/device.h>
#include <utils/assert.h>

#include ""
#include <hip/hip_runtime.h>
#include "kernels/gpu/cuda_context.h"
#include "core/device_context.h"
#include "utils/ctxmgr_lite.h"
#include "kernels/gpu/math_cublas.h"
#include "kernels/gpu/math_gpu.h"


namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void gpu_inner_prod_compute_run_kernel(int m, int n, int k, const T *A, const T *B, T *C) {
            __shared__ T ds_A[TRANS_BLOCK_DIM][TRANS_BLOCK_DIM];
            __shared__ T ds_B[TRANS_BLOCK_DIM][TRANS_BLOCK_DIM];

            int bx = blockIdx.x;
            int by = blockIdx.y;
            int tx = threadIdx.x;
            int ty = threadIdx.y;
            int Row = by * blockDim.y + ty;
            int Col = bx * blockDim.x + tx;

            T comp = T(0.f);
            T Cvalue = T(0.f);

            for (int t=0; t<(n - 1) / TRANS_BLOCK_DIM + 1; ++t) {
                if (Row < m && t * blockDim.x + tx < n)
                    ds_A[ty][tx] = A[Row*n+t*blockDim.x+tx];
                else
                    ds_A[ty][tx] = T(0.f);

                if (t * blockDim.y + ty < n && Col < k)
                    ds_B[ty][tx] = B[(t*blockDim.y + ty)*k+Col];
                else
                    ds_B[ty][tx] = T(0.f);

                __syncthreads();

                for (int i = 0; i < blockDim.x; ++i) {
                    //Cvalue += ds_A[ty][i] * ds_B[i][tx];
                    T t;
                    comp -= ds_A[ty][i] * ds_B[i][tx];
                    t = Cvalue - comp;
                    comp = (t - Cvalue) + comp;
                    Cvalue = t;
                }

                __syncthreads();

                if(Row < m && Col < k) {
                    C[Row*k+Col]=Cvalue;
                }
            }//end for
        
        
        }


        template<typename T>
        static void gpu_inner_prod_compute_run(const Tensor &lhs, const Tensor &rhs, bool transpose, Tensor &out) {
            const Shape &lhs_shape = lhs.sizes();
            const Shape &rhs_shape = rhs.sizes();

            const T *psrc = lhs.data<T>();
            const T *pdot = rhs.data<T>();
            T *pdst = out.data<T>();

#ifdef TS_USE_CUBLAS
            auto &context = ctx::ref<DeviceContext>();
            CUDAContextHandle* handle = reinterpret_cast<CUDAContextHandle*>(context.handle);
            auto cublas_handle = handle->cublas_handle();

            auto rhs_tranpose = transpose ? cublas::Trans : cublas::NoTrans;
            auto N = transpose ? rhs_shape[0] : rhs_shape[1];

            cublas::math<T>::gemm(cublas_handle, cublas::NoTrans, rhs_tranpose,
                lhs_shape[0], N, lhs_shape[1], T(1.f), psrc, pdot, T(0.f), pdst);
            /*cublas::math<T>::gemm(cublas_handle,cublas::RowMajor,cublas::NoTrans, cublas::NoTrans, 
                lhs_shape[0], rhs_shape[1], lhs_shape[1], 1,psrc, lhs_shape[1], pdot, rhs_shape[1], 0,pdst, rhs_shape[1]);*/
            
#else
            auto rhs_tranpose = transpose ? cublas::Trans : cublas::NoTrans;
            auto N = transpose ? rhs_shape[0] : rhs_shape[1];
            gpu::math<T>::gemm(
                    cublas::NoTrans, rhs_tranpose,
                    lhs_shape[0], N, lhs_shape[1], T(1.f), psrc, pdot, T(0.f), pdst);
            /*
            dim3 blocksize(CUDA_BLOCK(rhs_shape[1], TRANS_BLOCK_DIM), CUDA_BLOCK(lhs_shape[0], TRANS_BLOCK_DIM),1);
            dim3 threadsize(TRANS_BLOCK_DIM, TRANS_BLOCK_DIM,1);
            RUN_KERNEL(gpu_inner_prod_compute_run_kernel<T>, blocksize, threadsize, lhs_shape[0], lhs_shape[1], rhs_shape[1], psrc, pdot, pdst);
             */
#endif
        }

        void InnerProd::inner_prod(const Tensor &lhs, const Tensor &rhs, bool transpose, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_inner_prod_compute_run<TYPE>(lhs, rhs, transpose, out); break; }
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                break;
            }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(InnerProd, GPU, name::layer::inner_prod())
#ifdef TS_USE_CUDA_FP16
TS_REGISTER_FP16_OPERATOR(InnerProd, GPU, name::layer::inner_prod())
#endif
