#include "hip/hip_runtime.h"
//
// Created by kier on 2019/9/7.
//

#include "kernels/cpu/dragon/op_kernel.h"
#include "core/ieee754_float.h"

#include "kernels/common/third/dragon.h"

#ifdef TS_USE_CUDA_FP16
#include "kernels/gpu/cudax_fp16_math.h"
#endif
#include "kernels/gpu/gpu_kernel.h"

namespace ts {

    namespace dragon {

        namespace kernel {

/*! ROIAlign <T = float32, Device = CUDA> */

            template<typename T>
            __device__ T _ROIAlignInterpolate(
                    const T *Xdata,
                    const int H,
                    const int W,
                    T y,
                    T x) {
                if (y < -1.0 || y > H || x < -1.0 || x > W) return 0;
                if (y <= 0) y = 0;
                if (x <= 0) x = 0;

                int y_low = (int) y;
                int x_low = (int) x;
                int y_high;
                int x_high;

                if (y_low >= H - 1) {
                    y_high = y_low = H - 1;
                    y = (T) y_low;
                } else {
                    y_high = y_low + 1;
                }

                if (x_low >= W - 1) {
                    x_high = x_low = W - 1;
                    x = (T) x_low;
                } else {
                    x_high = x_low + 1;
                }

                T ly = y - y_low;
                T lx = x - x_low;
                T hy = 1. - ly, hx = 1. - lx;
#if __CUDA_ARCH__ >= 350
                T v1 = __ldg(Xdata + (y_low * W + x_low));
    T v2 = __ldg(Xdata + (y_low * W + x_high));
    T v3 = __ldg(Xdata + (y_high * W + x_low));
    T v4 = __ldg(Xdata + (y_high * W + x_high));
#else
                T v1 = Xdata[y_low * W + x_low];
                T v2 = Xdata[y_low * W + x_high];
                T v3 = Xdata[y_high * W + x_low];
                T v4 = Xdata[y_high * W + x_high];
#endif
                T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
                return w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4;
            }

            template<typename T>
            __global__ void _ROIAlign(
                    const int nthreads,
                    const int C,
                    const int H,
                    const int W,
                    const int pool_h,
                    const int pool_w,
                    const int sampling_ratio,
                    const float spatial_scale,
                    const T *Xdata,
                    const float *rois,
                    T *Ydata) {
                CUDA_1D_KERNEL_LOOP(y_idx, nthreads) {
                    int pw = y_idx % pool_w;
                    int ph = (y_idx / pool_w) % pool_h;
                    int c = (y_idx / pool_w / pool_h) % C;
                    int n = y_idx / pool_w / pool_h / C;

                    const T *offset_rois = rois + n * 5;
                    int roi_batch_ind = offset_rois[0];

                    if (roi_batch_ind < 0) {
                        Ydata[y_idx] = 0;
                        continue;
                    }

                    T roi_start_w = offset_rois[1] * spatial_scale;
                    T roi_start_h = offset_rois[2] * spatial_scale;
                    T roi_end_w = offset_rois[3] * spatial_scale;
                    T roi_end_h = offset_rois[4] * spatial_scale;

                    T roi_width = max(roi_end_w - roi_start_w, (T) 1.);
                    T roi_height = max(roi_end_h - roi_start_h, (T) 1.);
                    T bin_size_h = (T) roi_height / (T) pool_h;
                    T bin_size_w = (T) roi_width / (T) pool_w;

                    const T *offset_Xdata = Xdata + (roi_batch_ind * C + c) * H * W;

                    int roi_bin_grid_h = (sampling_ratio > 0) ?
                                         sampling_ratio : ceil(roi_height / pool_h);
                    int roi_bin_grid_w = (sampling_ratio > 0) ?
                                         sampling_ratio : ceil(roi_width / pool_w);

                    T output_val = 0.;
                    const T num_bin_grids = roi_bin_grid_h * roi_bin_grid_w;

                    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
                        const T y = roi_start_h + ph * bin_size_h +
                                    static_cast<T>(iy + .5f) * bin_size_h /
                                    static_cast<T>(roi_bin_grid_h);
                        for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                            const T x = roi_start_w + pw * bin_size_w +
                                        static_cast<T>(ix + .5f) * bin_size_w /
                                        static_cast<T>(roi_bin_grid_w);
                            output_val += _ROIAlignInterpolate(
                                    offset_Xdata, H, W, y, x);
                        }
                    }
                    output_val /= num_bin_grids;
                    Ydata[y_idx] = output_val;
                }
            }

            template<>
            void ROIAlign<float, HIPContext>(
                    const int C,
                    const int H,
                    const int W,
                    const int pool_h,
                    const int pool_w,
                    const int num_rois,
                    const float spatial_scale,
                    const int sampling_ratio,
                    const float *x,
                    const float *rois,
                    float *y,
                    HIPContext *ctx) {
                auto nthreads = num_rois * C * pool_h * pool_w;
                RUN_KERNEL_STREAM(_ROIAlign<float>,
                                  CUDA_BLOCKS(nthreads), CUDA_THREADS,
                                  0, ctx->hip_stream(),
                                  nthreads, C, H, W, pool_h, pool_w,
                                  sampling_ratio, spatial_scale, x, rois, y);
            }

#ifdef TS_USE_CUDA_FP16

/*! ROIAlign <T = float16, Device = CUDA> */

            __device__ float _ROIAlignInterpolate(
                    const half *Xdata,
                    const int H,
                    const int W,
                    float y,
                    float x) {
                if (y < -1.0 || y > H || x < -1.0 || x > W) return 0.f;
#if __CUDA_ARCH__ >= 530
                if (y <= 0) y = 0;
                if (x <= 0) x = 0;

                int y_low = (int)y;
                int x_low = (int)x;
                int y_high;
                int x_high;

                if (y_low >= H - 1) {
                    y_high = y_low = H - 1;
                    y = (float)y_low;
                } else {
                    y_high = y_low + 1;
                }

                if (x_low >= W - 1) {
                    x_high = x_low = W - 1;
                    x = (float)x_low;
                } else {
                    x_high = x_low + 1;
                }

                const float ly = y - y_low;
                const float lx = x - x_low;
                const float hy = 1. - ly, hx = 1. - lx;
                const float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
                const float v1 = __half2float(__ldg(Xdata + (y_low * W + x_low)));
                const float v2 = __half2float(__ldg(Xdata + (y_low * W + x_high)));
                const float v3 = __half2float(__ldg(Xdata + (y_high * W + x_low)));
                const float v4 = __half2float(__ldg(Xdata + (y_high * W + x_high)));
                const float value = w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4;
#else
                const float value = 0.f;
#endif
                return value;
            }

            __global__ void _ROIAlignHalf(
                    const int nthreads,
                    const int C,
                    const int H,
                    const int W,
                    const int pool_h,
                    const int pool_w,
                    const int sampling_ratio,
                    const float spatial_scale,
                    const half *Xdata,
                    const float *rois,
                    half *Ydata) {
                CUDA_1D_KERNEL_LOOP(y_idx, nthreads) {
#if __CUDA_ARCH__ >= 530
                    int pw = y_idx % pool_w;
                    int ph = (y_idx / pool_w) % pool_h;
                    int c = (y_idx / pool_w / pool_h) % C;
                    int n = y_idx / pool_w / pool_h / C;

                    const float* offset_rois = rois + n * 5;
                    int roi_batch_ind = offset_rois[0];

                    if (roi_batch_ind < 0) {
                        Ydata[y_idx] = __float2half(0.f);
                        continue;
                    }

                    float roi_start_w = offset_rois[1] * spatial_scale;
                    float roi_start_h = offset_rois[2] * spatial_scale;
                    float roi_end_w = offset_rois[3] * spatial_scale;
                    float roi_end_h = offset_rois[4] * spatial_scale;

                    float roi_width = max(roi_end_w - roi_start_w, 1.f);
                    float roi_height = max(roi_end_h - roi_start_h, 1.f);
                    float bin_size_h = (float)roi_height / (float)pool_h;
                    float bin_size_w = (float)roi_width / (float)pool_w;

                    const half* offset_Xdata = Xdata + (roi_batch_ind * C + c) * H * W;

                    int roi_bin_grid_h = (sampling_ratio > 0) ?
                        sampling_ratio : ceil(roi_height / pool_h);
                    int roi_bin_grid_w = (sampling_ratio > 0) ?
                        sampling_ratio : ceil(roi_width / pool_w);

                    float output_val = 0.;
                    const float num_bin_grids = roi_bin_grid_h * roi_bin_grid_w;

                    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
                        const float y = roi_start_h + ph * bin_size_h +
                            static_cast<float>(iy + .5f) * bin_size_h /
                                static_cast<float>(roi_bin_grid_h);
                        for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                            const float x = roi_start_w + pw * bin_size_w +
                                static_cast<float>(ix + .5f) * bin_size_w /
                                    static_cast<float>(roi_bin_grid_w);
                            output_val += _ROIAlignInterpolate(
                                offset_Xdata, H, W, y, x);
                        }
                    }
                    output_val /= num_bin_grids;
                    Ydata[y_idx] = __float2half(output_val);
#endif
                }
            }

            template<>
            void ROIAlign<float16, HIPContext>(
                    const int C,
                    const int H,
                    const int W,
                    const int pool_h,
                    const int pool_w,
                    const int num_rois,
                    const float spatial_scale,
                    const int sampling_ratio,
                    const float16 *x,
                    const float *rois,
                    float16 *y,
                    HIPContext *ctx) {
                auto nthreads = num_rois * C * pool_h * pool_w;
                RUN_KERNEL_STREAM(_ROIAlignHalf,
                                  CUDA_BLOCKS(nthreads), CUDA_THREADS,
                                  0, ctx->hip_stream(),
                                  nthreads, C, H, W, pool_h, pool_w,
                                  sampling_ratio, spatial_scale,
                                  reinterpret_cast<const half *>(x), rois,
                                  reinterpret_cast<half *>(y));
            }
#endif
        }  // namespace kernel

    }  // namespace dragon

} // namespace ts