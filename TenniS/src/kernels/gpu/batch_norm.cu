#include "hip/hip_runtime.h"
#include <kernels/gpu/batch_norm.h>
#include <core/tensor_builder.h>

#include <global/operator_factory.h>
#include "global/fp16_operator_factory.h"
#include <backend/name.h>
#include <utils/assert.h>
#include <core/device.h>
#include <vector>

#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include <runtime/runtime.h>

#include "kernels/gpu/gpu_kernel.h"

namespace ts {
    namespace gpu {
        template<typename T>
        static __global__ void gpu_batch_norm_compute_kernel(const T* data, T* out, int size, int step, int slice,
                                        const T* mean, const T* variance ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                out[index] = (data[index] - mean[dim]) * variance[dim];
            }
        }

        template<typename T>
        static __global__ void vec_kernel(const int N, T one, T epsilon, const T* input,T* output) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;

            for (; index < N; index += blockDim.x * gridDim.x) {
                output[index] = one / sqrt(input[index] + epsilon);
            }
        }

#ifdef TS_USE_CUDA_FP16
        template<>
        __global__ void vec_kernel<half>(const int N, half one, half epsilon, const half* input, half* output) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            for (; index < N; index += blockDim.x * gridDim.x) {
                output[index] = one / hsqrt(input[index] + epsilon);
            }
        }
#endif

        template<typename T>
        static void gpu_batch_norm_compute_run(const Tensor &x, const Tensor &mean,
                                               const Tensor &variance, int dim, float epsilon, Tensor &out) {
            const Shape &shape = x.sizes();
            //int predims = 1;
            int backdims = 1;
            //for (int i = 0; i < dim; i++) {
            //    predims *= shape[i];
            //}

            for (int i = dim + 1; i < shape.size(); i++) {
                backdims *= shape[i];
            }

            const T *psrc = x.data<T>();
            const T *pmean = mean.data<T>();
            const T *pvariance = variance.data<T>();
            T *pdst = out.data<T>();

            Shape vec_shape = variance.sizes();
            Tensor vec_tensor(RuntimeContext::FlowMemory(), variance.dtype(), vec_shape, MemoryDevice(variance.device()));
            T* vec_data = vec_tensor.data<T>();

            int vec_len = vec_tensor.count();
            dim3 block_size(CUDA_THREAD_NUM);
            dim3 grid_size(CUDA_BLOCK(vec_len, block_size.x));
            T one(1.f);
            T temp_epsilon(epsilon);
            RUN_KERNEL(vec_kernel<T>, grid_size, block_size, vec_len, one, temp_epsilon, pvariance,vec_data);

            RUN_KERNEL(gpu_batch_norm_compute_kernel<T>, CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM,
                       psrc, pdst, out.count(), backdims, shape[dim], pmean, vec_data);
        }


        void BatchNorm::batch_norm(const Tensor &x, const Tensor &mean, const Tensor &variance,
                                   int dim, float epsilon, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_batch_norm_compute_run<TYPE>(x, mean, variance, dim, epsilon, out); break; }
                //DECLARE_COMPUTE_RUN(INT8, int8_t);
                //DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                //DECLARE_COMPUTE_RUN(INT16, int16_t);
                //DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                //DECLARE_COMPUTE_RUN(INT32, int32_t);
                //DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                //DECLARE_COMPUTE_RUN(INT64, int64_t);
                //DECLARE_COMPUTE_RUN(UINT64, uint64_t);
#ifdef TS_USE_CUDA_FP16
                DECLARE_COMPUTE_RUN(FLOAT16, half);
#endif
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support data type(" << dtype << "): " << type_str(dtype) << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(BatchNorm, GPU, name::layer::batch_norm())
#ifdef TS_USE_CUDA_FP16
TS_REGISTER_FP16_OPERATOR(BatchNorm, ts::GPU, name::layer::batch_norm())
#endif
